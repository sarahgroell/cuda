#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "MandelbrotMath.h"

#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"

using namespace gpu;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/
__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h,DomaineMath domainMath,uint n,float t);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, DomaineMath domainMath,uint n, float t)
    {

    MandelbrotMath mandelbrotMath = MandelbrotMath(n); // ici pour preparer cuda

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;

    double x;
    double y;

    int i;
    int j;

    int s = TID; // in [0,...
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[

	domainMath.toXY(i, j, &x, &y);
	mandelbrotMath.colorXY(&ptrDevPixels[s], x, y, t);

	s += NB_THREAD;
	}

    }
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * i in [0,h[
 * j in [0,w[
 *
 * code commun a:
 * 	- entrelacementOMP
 * 	- forAutoOMP
 */
//void workPixel(uchar4* ptrColorIJ, int i, int j, const DomaineMath& domaineMath, MandelbrotMath* ptrMandelbrotMath,float t)
//    {
//    // (i,j) domaine ecran dans N2
//    // (x,y) domaine math dans R2
//
//    double x;
//    double y;
//    domaineMath.toXY(i, j, &x, &y); // fill (x,y) from (i,j)
//
//    // float t=variateurAnimation.get();
//
//    ptrMandelbrotMath->colorXY(ptrColorIJ, x, y, t); // in [01]
//    }
/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
