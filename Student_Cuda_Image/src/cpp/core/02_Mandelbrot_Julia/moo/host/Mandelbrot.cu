#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include "Mandelbrot.h"
#include <assert.h>

#include "DomaineMath_GPU.h"
#include "IndiceTools_GPU.h"

using namespace gpu;
using std::cout;
using std::endl;
/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/
extern __global__ void mandelbrot(uchar4* ptrDevPixels,uint w, uint h, DomaineMath domaineMath,uint n,float dt);

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/
Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, float dt, uint n, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_Julia_CUDA_RGBA_uchar4", domaineMath), variateurAnimation(Interval<float>(30, 100), dt)
    {
    this->n = n;
    // Inputs
//    this->dt = 0;

// Tools
    this->t = 0; // protected dans Animable
    }

Mandelbrot::~Mandelbrot()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    //Device::lastCudaError("fractale rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    mandelbrot<<<dg,db>>>(ptrDevPixels,w,h,domaineMath,n,t);
    // le kernel est importer ci-dessus (ligne 19)

    //Device::lastCudaError("fractale rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void Mandelbrot::animationStep()
    {
//    t += n;
    n = variateurAnimation.varierAndGet();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

