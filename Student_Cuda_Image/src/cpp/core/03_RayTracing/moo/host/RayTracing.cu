#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include "RayTracing.h"
#include "SphereCreator.h"
#include <assert.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void rayTracing(uchar4* ptrDevPixels, Sphere* ptrDevTabSphere, int nbSphere,uint w, uint h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "RayTracing_Cuda_RGBA_uchar4")
    {

    // Inputs
    this->dt = dt;
    this->nbSphere = 35;

    SphereCreator shereCreator(this->nbSphere, w, h); // sur la pile
    Sphere* ptrTabSphere = shereCreator.getTabSphere();

    this->sizeOctetSphere = sizeof(Sphere) * this->nbSphere;

    // transfert to GM
    toGM(ptrTabSphere);
    // transfert to CM
    toCM(ptrTabSphere); // a implemneter
    // Tools
    this->t = 0; // protected dans Animable
    }

RayTracing::~RayTracing()
    {
    // rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    //Device::lastCudaError("RayTracing rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // TODO lancer le kernel avec <<<dg,db>>>
    // le kernel est importer ci-dessus (ligne 19)

    rayTracing<<<dg,db>>>(ptrDevPixels,this->ptrDevTabSphere, this->nbSphere,w, h, t);

    //Device::lastCudaError("RayTracing rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

void RayTracing::toGM(Sphere* ptrSphere)
    {
    Device::malloc(&this->ptrDevTabSphere, sizeOctetSphere);

    Device::memcpyHToD(this->ptrDevTabSphere, ptrSphere, sizeOctetSphere);
    }

void RayTracing::toCM(Sphere* ptrDevSphereCreator)
    {

    }
/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
